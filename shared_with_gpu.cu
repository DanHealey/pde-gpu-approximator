#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <cmath>
#include <iostream>
#include <algorithm>
#include <sys/time.h>

__device__ double pi = 3.14159265358979323846;
__device__ constexpr double n1 = 2;
__device__ constexpr double m1 = 2;
__device__ constexpr double k1 = 2;

long long time_diff(struct timeval start, struct timeval end) {
    return (end.tv_sec - start.tv_sec) * 1000000LL + (end.tv_usec - start.tv_usec);
}

__device__ double exact_phi(double x, double y, double z) {
    return sin(n1 * pi * x) * cos(m1 * pi * y) * sin(k1 * pi * z);
}

__device__ double f(double x, double y, double z) {
    return -(k1 * k1 + m1 * m1 + n1 * n1) * (pi * pi) * exact_phi(x, y, z);
}

__global__ void initialize(double* phi, double* phi_old, double* phi_actual, double* f_phi, size_t N, double h) {
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    size_t j = blockIdx.y * blockDim.y + threadIdx.y;
    size_t k = blockIdx.z * blockDim.z + threadIdx.z;

    if (i >= 0 && j>=0 && k>=0 && i < N && j < N && k < N) {
        double x = i * h;
        double y = j * h;
        double z = k * h;
        size_t idx = i + j * N + k * N * N;

        phi[idx] = 0.0;
        phi_old[idx] = 0.0;
        phi_actual[idx] = exact_phi(x, y, z);
        f_phi[idx] = f(x, y, z);

        if (i == 0 || i == N - 1 || j == 0 || j == N - 1 || k == 0 || k == N - 1) {
            phi[idx] = exact_phi(x, y, z);
            phi_old[idx] = phi[idx];
        }
    }
}

__global__ void update_phi(double* phi, double* phi_old, 
    double* f_phi, size_t N, double h) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int k = blockIdx.z * blockDim.z + threadIdx.z;

    if (i > 0 && i < N - 1 && j > 0 
        && j < N - 1 && k > 0 && k < N - 1) {
        size_t idx = i + j * N + k * N * N;
        phi[idx] = (
            phi_old[(i - 1) + j * N + k * N * N] +
            phi_old[(i + 1) + j * N + k * N * N] +
            phi_old[i + (j - 1) * N + k * N * N] +
            phi_old[i + (j + 1) * N + k * N * N] +
            phi_old[i + j * N + (k - 1) * N * N] +
            phi_old[i + j * N + (k + 1) * N * N] -
            f_phi[idx] * (h * h)
        ) / 6.0;
    }
}

__global__ void compute_error_and_convergence(double* phi, double* phi_old, double* phi_actual, size_t N, double* error, double *conv) {
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    size_t j = blockIdx.y * blockDim.y + threadIdx.y;
    size_t k = blockIdx.z * blockDim.z + threadIdx.z;

    if (i > 0 && i < N - 1 && j > 0 && j < N - 1 && k > 0 && k < N - 1) {
        size_t idx = i + j * N + k * N * N;

        // Calculate error
        double diff = phi[idx] - phi_actual[idx];
        atomicAdd(error, diff * diff);

        // Calculate convergence
        diff = phi[idx] - phi_old[idx];
        atomicAdd(conv, diff * diff);
    }
}

hipError_t GPU_ERROR;

void finite_difference() {
    // METRICS
    long long min_iteration_time, max_iteration_time, total_time = 0, avg_iteration_time;
    struct timeval start_time, end_time, start_iter_time, end_iter_time;

    gettimeofday(&start_time, NULL);

    const size_t N = 15;
    const double h = 1.0 / (N - 1);
    const double tol = 1e-6;

    size_t size = N * N * N * sizeof(double);
    double *d_phi, *d_phi_old, *d_phi_actual, *d_f_phi, *d_error, *d_conv;

    double phi_actual[N * N * N]; // phi(x, y, z)
    double f_phi[N * N * N]; // f(x, y, z)
    double phi[N * N * N]; // intermediate "new" phi(x, y, z)
    double phi_old[N * N * N]; // intermediate "old" phi(x, y, z)

    GPU_ERROR = hipMalloc(&d_phi, size);
    GPU_ERROR = hipMalloc(&d_phi_old, size);
    GPU_ERROR = hipMalloc(&d_phi_actual, size);
    GPU_ERROR = hipMalloc(&d_f_phi, size);
    GPU_ERROR = hipMalloc(&d_error, sizeof(double));
    GPU_ERROR = hipMalloc(&d_conv, sizeof(double));

    dim3 threadsPerBlock(8, 8, 8);
    dim3 numBlocks((N + threadsPerBlock.x - 1) / threadsPerBlock.x,
                   (N + threadsPerBlock.y - 1) / threadsPerBlock.y,
                   (N + threadsPerBlock.z - 1) / threadsPerBlock.z);

    initialize<<<numBlocks, threadsPerBlock>>>(d_phi, d_phi_old, d_phi_actual, d_f_phi, N, h);

    double error = INFINITY;
    double conv = INFINITY;
    int iteration = 1;

    do {
        gettimeofday(&start_iter_time, NULL);

        update_phi<<<numBlocks, threadsPerBlock>>>(d_phi, d_phi_old, d_f_phi, N, h);
        GPU_ERROR = hipDeviceSynchronize();

        error = 0.0;
        conv = 0.0;
        GPU_ERROR = hipMemset(d_error, 0, sizeof(double));
        GPU_ERROR = hipMemset(d_conv, 0, sizeof(double));
        compute_error_and_convergence<<<numBlocks, threadsPerBlock>>>(d_phi, d_phi_old, d_phi_actual, N, d_error, d_conv);
        GPU_ERROR = hipDeviceSynchronize();
        GPU_ERROR = hipMemcpy(d_phi_old, d_phi, size, hipMemcpyDeviceToDevice);
        GPU_ERROR = hipMemcpy(&error, d_error, sizeof(double), hipMemcpyDeviceToHost);
        GPU_ERROR = hipMemcpy(&conv, d_conv, sizeof(double), hipMemcpyDeviceToHost);

        // Start tracking from 10th iteration
        if (iteration >= 10) {
            gettimeofday(&end_iter_time, NULL);
            long long iteration_time = time_diff(start_iter_time, end_iter_time);
            if (iteration == 10) {
                min_iteration_time = iteration_time;
                max_iteration_time = iteration_time;
            } else {
                min_iteration_time = std::min(min_iteration_time, iteration_time);
                max_iteration_time = std::max(max_iteration_time, iteration_time);
            }
            total_time += iteration_time;
            avg_iteration_time = total_time / (iteration - 9);
        }

        // printf("Iteration %d: Max Time: %lld us, Min Time: %lld us, Avg Time: %lld us\n", 
        //     iter, 
        //     max_iteration_time, 
        //     min_iteration_time, 
        //     avg_iteration_time
        // );
        //std::cout << "Error: " << error << std::endl;
        //std::cout << "Convergence: " << conv << std::endl;

        iteration++;
    } while (conv > tol);

    GPU_ERROR = hipMemcpy(phi, d_phi, size, hipMemcpyDeviceToHost);

    GPU_ERROR = hipFree(d_phi);
    GPU_ERROR = hipFree(d_phi_old);
    GPU_ERROR = hipFree(d_phi_actual);
    GPU_ERROR = hipFree(d_f_phi);
    GPU_ERROR = hipFree(d_error);

    gettimeofday(&end_time, NULL);

    total_time = time_diff(start_time, end_time);
    
    printf("[FINAL RESULT]\n");
    printf("Total computation time: %lld us\n", total_time);
    printf("Average iteration time: %lld us\n", avg_iteration_time);
    printf("Minimum iteration time: %lld us\n", min_iteration_time);
    printf("Maximum iteration time: %lld us\n", max_iteration_time);
    printf("Iterations: %d\n", iteration);
    printf("Error: %f\n", error);

}

int main() {
    finite_difference();
    return 0;
}
